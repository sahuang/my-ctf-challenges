#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "calc.h"
#include <stdlib.h>

using namespace std;

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {

    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}


void matrixMultiplication(float* A, float* B, float* C, int N) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    if (N * N > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(N) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(N) / double(threadsPerBlock.y));
    }

    matrixMultiplicationKernel << <blocksPerGrid, threadsPerBlock >> > (A, B, C, N);
}