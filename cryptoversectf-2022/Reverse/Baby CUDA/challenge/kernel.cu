﻿#include <iostream>
#include <vector>
#include <string>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "calc.h"
#include "dev_array.h"
#include <math.h>

using namespace std;

int main()
{
    // Perform matrix multiplication C = A*B
    // where A, B and C are NxN matrices
    int N = 4;
    int SIZE = N * N;

    // Allocate memory on the host
    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    // Read flag
    string flag;
    cout << "Enter flag: ";
    cin >> flag;
    if (flag.length() != SIZE) {
        exit(1);
    }

    // Initialize matrices on the host
    int curr = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_A[i * N + j] = flag[curr];
            h_B[i * N + j] = curr++;
        }
    }

    h_B[0]++;
    h_B[3]+=2;
    h_B[5] += 2;
    h_B[6] += 3;
    h_B[10] += 1;
    h_B[14] += 5;

    // Allocate memory on the device
    dev_array<float> d_A(SIZE);
    dev_array<float> d_B(SIZE);
    dev_array<float> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);

    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();

    vector<vector<int>> res{ 
        {2755,3324,4553,4150},
        {2534,3087,4271,3863},
        {1828,2145,2903,2739},
        {2436,2830,3926,3490} };

    // cvctf{CuD4_B@@M}
    for (int ROW = 0; ROW < N; ROW++) {
        for (int COL = 0; COL < N; COL++) {
            if ((int)h_C[ROW * N + COL] != res[ROW][COL]) exit(1);
        }
    }

    cout << "Congratulations!\n";

    return 0;
}